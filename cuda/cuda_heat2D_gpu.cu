#include "hip/hip_runtime.h"
#include "lcutil.h"
#include "timestamp.h"

/* 

	Declarations
	
*/


struct Parms { 
  float cx;
  float cy;
} parms = {0.1, 0.1};

/* 

	GPU functions
	
*/

__global__ void iterator_gpu(const float * __restrict__ T_source, float * __restrict__ T_destination, const int NXPROB, const int NYPROB,struct Parms parms){

	const int ix = blockIdx.x * blockDim.x + threadIdx.x ;
	const int iy = blockIdx.y * blockDim.y + threadIdx.y ;
	
	
	if (ix>0 && ix<NXPROB-1 && iy>0 && iy<NYPROB-1)
		*(T_destination+ix*NYPROB+iy) = *(T_source+ix*NYPROB+iy)  + 
                          			parms.cx * (*(T_source+(ix+1)*NYPROB+iy) +
                          			*(T_source+(ix-1)*NYPROB+iy) - 
                          			2.0 * *(T_source+ix*NYPROB+iy)) +
                          			parms.cy * (*(T_source+ix*NYPROB+iy+1) +
                         			*(T_source+ix*NYPROB+iy-1) - 
                          			2.0 * *(T_source+ix*NYPROB+iy));
	

}

extern float Iterator_GPU(float* u, int NXPROB, int NYPROB, int BLOCK_SIZE_X, int BLOCK_SIZE_Y, int STEPS){

	// --- GPU temperature distribution
	float *d_u_z_0;
	CUDA_SAFE_CALL( hipMalloc((void**)&d_u_z_0,  NXPROB * NYPROB * sizeof(float)) );
	float *d_u_z_1; 
	CUDA_SAFE_CALL( hipMalloc((void**)&d_u_z_1,  NXPROB * NYPROB * sizeof(float)) );
	
	// --- Grid size
	dim3 dimBlock(BLOCK_SIZE_X, BLOCK_SIZE_Y);
	dim3 dimGrid (FRACTION_CEILING(NXPROB, BLOCK_SIZE_X), FRACTION_CEILING(NYPROB, BLOCK_SIZE_Y));
	
	hipEvent_t start, stop;
	CUDA_SAFE_CALL( hipEventCreate(&start) );
	CUDA_SAFE_CALL( hipEventCreate(&stop) );
	
	
	CUDA_SAFE_CALL( hipMemcpy(d_u_z_0, u, NXPROB * NYPROB * sizeof(float), hipMemcpyHostToDevice) );
	CUDA_SAFE_CALL( hipMemcpy(d_u_z_1, u+NXPROB*NYPROB, NXPROB * NYPROB * sizeof(float), hipMemcpyHostToDevice) );
	
	
	CUDA_SAFE_CALL( hipEventRecord(start) );
	// --- Jacobi iterations on the device
	for (int it=0; it<STEPS; it=it+2) {
		iterator_gpu<<<dimGrid, dimBlock>>>(d_u_z_0, d_u_z_1, NXPROB, NYPROB, parms);   // --- Update d_u_z_1     starting from data stored in d_u_z_0
		iterator_gpu<<<dimGrid, dimBlock>>>(d_u_z_1, d_u_z_0, NXPROB, NYPROB, parms);   // --- Update d_u_z_0     starting from data stored in d_u_z_1
	}      
	CUDA_SAFE_CALL( hipEventRecord(stop) );

	// --- Copy result from device to host
	CUDA_SAFE_CALL( hipMemcpy(u, d_u_z_0, NXPROB * NYPROB * sizeof(float), hipMemcpyDeviceToHost) );
	CUDA_SAFE_CALL( hipEventSynchronize(stop) );
	float msecs = 0;
	CUDA_SAFE_CALL( hipEventElapsedTime(&msecs, start, stop) );
	
	// --- Release device memory
      	CUDA_SAFE_CALL(hipFree(d_u_z_0));
      	CUDA_SAFE_CALL(hipFree(d_u_z_1));
      	
      	return msecs;

}

